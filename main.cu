
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <ctime>
#include <string>

//add values of array x to array y
__global__ void add(int n, float *x, float *y)
{

    for (int i = 0; i < n; i++)
    {
            y[i] += x[i];
    }
}

__global__ void block_add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main() {
    int N = 1048576;

    float *x;
    float *y;
    hipMallocManaged(&x, N* sizeof(float));
    hipMallocManaged(&y, N* sizeof(float));

    //init array x and y on host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    int blockSize = 512*2048;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout << numBlocks << std::endl;
    auto start = std::chrono::system_clock::now();
    //add<<<1, 1>>>(N, x, y);
    block_add<<<numBlocks, blockSize>>>(N, x, y);
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();

    //Check for errors
    float maxError = 0.0f;
    for(int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    std::cout << "MaxError: " << maxError << std::endl;
    std::chrono::duration<double> elapsed_seconds = end-start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);
    std::cout << "Time elapsed: " << elapsed_seconds.count() << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}
